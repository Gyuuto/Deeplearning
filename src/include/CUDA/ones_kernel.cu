
#include <hip/hip_runtime.h>
__global__
void ones ( int m, int n, float* A ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  if( i < m && j < n ) {
    A[i*n + j] = 1.0f;
  }
}

void cuda_ones_kernel ( int m, int n, float* A ) {
  const int thread_num = 32;

  dim3 grid(16, 16, 1);
  dim3 block(thread_num, thread_num, 1);
  
  ones<<<grid, block>>>( m, n, A );
}